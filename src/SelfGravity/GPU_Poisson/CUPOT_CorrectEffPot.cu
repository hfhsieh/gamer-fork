#include "hip/hip_runtime.h"
#include "CUPOT.h"
#include <stdio.h>

#if ( defined GRAVITY  &&  defined GREP )


// external functions and GPU-related set-up
#ifdef __HIPCC__

// variables reside in constant memory
__constant__ double c_GREP_Data  [GR_POT_NAUX_MAX];
__constant__ double c_GREP_Radius[GR_POT_NAUX_MAX];
__constant__ double c_GREP_Center[3];
__constant__ int    c_GREP_NBin;


//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_SetConstMem_GREffPot
// Description :  Set the constant memory used by CUPOT_CorrectEffPot()
//
// Note        :  1. Adopt the suggested approach for CUDA version >= 5.0
//                2. Invoked by CUAPI_Init_GREffPot()
//
// Parameter   :  None
//
// Return      :  0/-1 : successful/failed
//---------------------------------------------------------------------------------------------------
__host__
int CUPOT_SetConstMem_GREffPot( double h_GREP_Data[], double h_GREP_Radius[], double h_GREP_Center[],
                                int    h_GREP_NBin )
{

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Data),   h_GREP_Data,   GR_POT_NAUX_MAX*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -1;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Radius), h_GREP_Radius, GR_POT_NAUX_MAX*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -2;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Center), h_GREP_Center,               3*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -3;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_NBin),  &h_GREP_NBin,                   sizeof(int),
                                            0, hipMemcpyHostToDevice)  )
      return -4;

   return 0;

} // FUNCTION : CUPOT_SetConstMem_GREffPot

#endif // ifdef __HIPCC__


#define LinearInterp( x, xa, xb, ya, yb )   ( ( ((x) - (xa)) * (yb) + ((xb) - (x)) * (ya) ) / ((xb) - (xa)) )




//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_CorrectEffPot
// Description :  Do/Undo the GR potential correction
//
// Note        :  1. Support CPU only in current version
//                2. The potential correction calculated at the current step is applied to
//                   both g_Pot_Array_New and g_Pot_Array_USG in current version
//
// Parameter   :  g_Pot_Array_New   : Array storing the input potential (at the current step)
//                                    --> _New: to be distinguishable from g_Pot_Array_USG[], which is defined at the previous step
//                g_Pot_Array_USG   : Array storing the input potential for UNSPLIT_GRAVITY (at the previous step)
//                g_Corner_Array    : Array storing the physical corner coordinates of each patch
//                dh                : Cell size
//                Undo              : Add (true) or subtract (false) potential correction to the input potential
//                USG               : Flag to indicate which potential is input
//
//-------------------------------------------------------------------------------------------------------
__global__
void CUPOT_CorrectEffPot(       real   g_Pot_Array_New[][ CUBE(GRA_NXT) ],
                                real   g_Pot_Array_USG[][ CUBE(USG_NXT_G) ],
                          const double g_Corner_Array [][3],
                          const real dh, const bool Undo, const bool USG )
{

// declare index for loop
#  ifdef UNSPLIT_GRAVITY
   const int IDX    = ( USG ) ? USG_NXT_G      : GRA_NXT;
   const int IDX_GZ = ( USG ) ? USG_GHOST_SIZE : GRA_GHOST_SIZE;
#  else
   const int IDX    = GRA_NXT;
   const int IDX_GZ = GRA_GHOST_SIZE;
#  endif

   const int IDX_sqr = SQR (IDX);

   const int P = blockIdx.x;
   {
//    loop over all cells of the target patch
//    _g0: indices for the arrays without any ghost zone
      CGPU_LOOP( idx_g0, CUBE(IDX) )
      {

         const int i_g0 = idx_g0 % IDX;
         const int j_g0 = idx_g0 % IDX_sqr / IDX;
         const int k_g0 = idx_g0 / IDX_sqr;

         const double dx = g_Corner_Array[P][0] + (double)((i_g0-IDX_GZ)*dh) - c_GREP_Center[0];
         const double dy = g_Corner_Array[P][1] + (double)((j_g0-IDX_GZ)*dh) - c_GREP_Center[1];
         const double dz = g_Corner_Array[P][2] + (double)((k_g0-IDX_GZ)*dh) - c_GREP_Center[2];

         const double r  = SQRT( SQR(dx) + SQR(dy) + SQR(dz) );


         double phi;

         if ( r < c_GREP_Radius[0] )
         {
            phi = c_GREP_Data[0];
         }

         else if ( r < c_GREP_Radius[c_GREP_NBin-1] )
         {
//          if empty bins are removed, the separations between bins are not equal in linear/logarithmic scale
//          use binary search algorithm to find the index of bin
            int Idx, Min = 0, Max = c_GREP_NBin-1;

            while (  ( Idx=(Min+Max)/2 ) != Min  )
            {
               if   ( c_GREP_Radius[Idx] > r )  Max = Idx;
               else                             Min = Idx;
            }

            phi = LinearInterp( r, c_GREP_Radius[Idx], c_GREP_Radius[Idx+1], c_GREP_Data[Idx], c_GREP_Data[Idx+1] );
         }

         else
         {
            phi = c_GREP_Data[c_GREP_NBin-1];
         }


         if ( Undo )   phi = -phi;

#        ifdef UNSPLIT_GRAVITY
         if ( USG )
            g_Pot_Array_USG[P][idx_g0] += (real)phi;
         else
            g_Pot_Array_New[P][idx_g0] += (real)phi;
#        else
            g_Pot_Array_New[P][idx_g0] += (real)phi;
#        endif
      } // CGPU_LOOP( idx_g0, CUBE(PS1) )
   } // for (int P=0; P<NPatchGroup*8; P++)

} // FUNCTION : CPU_CorrectEffPot


#endif // #if ( defined GRAVITY  &&  defined GREP )
