#include "hip/hip_runtime.h"
#include "CUPOT.h"
#include <stdio.h>

#if ( defined GRAVITY  &&  defined GREP )


// external functions and GPU-related set-up
#ifdef __HIPCC__

// variables reside in constant memory
__constant__ double c_GREP_Data[GR_POT_NAUX_MAX];
__constant__ double c_GREP_Edge[GR_POT_NAUX_MAX];
__constant__ double c_GREP_Center[3];
__constant__ double c_r_max2;
__constant__ int    c_GREP_NBin;


//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_SetConstMem_GREffPot
// Description :  Set the constant memory used by CUPOT_CorrectEffPot()
//
// Note        :  1. Adopt the suggested approach for CUDA version >= 5.0
//                2. Invoked by CUAPI_Init_GREffPot()
//
// Parameter   :  None
//
// Return      :  0/-1 : successful/failed
//---------------------------------------------------------------------------------------------------
__host__
int CUPOT_SetConstMem_GREffPot( double h_GREP_Data[], double h_GREP_Edge[], double h_GREP_Center[],
                                double h_r_max2, int h_GREP_NBin )
{

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Data),   h_GREP_Data,   GR_POT_NAUX_MAX*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -1;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Edge),   h_GREP_Edge,   GR_POT_NAUX_MAX*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -2;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_Center), h_GREP_Center,               3*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -3;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_r_max2),     &h_r_max2,                      sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -4;

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( c_GREP_NBin),  &h_GREP_NBin,                   sizeof(int),
                                            0, hipMemcpyHostToDevice)  )
      return -5;

   return 0;

} // FUNCTION : CUPOT_SetConstMem_GREffPot

#endif // ifdef __HIPCC__


#define LinearInterp( x, xa, xb, ya, yb )   ( ( ((x) - (xa)) * (yb) + ((xb) - (x)) * (ya) ) / ((xb) - (xa)) )




//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_CorrectEffPot
// Description :  Do/Undo the GR potential correction
//
// Note        :  1. Support CPU only in current version
//                2. The potential correction calculated at the current step is applied to
//                   both g_Pot_Array_New and g_Pot_Array_USG in current version
//
// Parameter   :  g_Pot_Array_New   : Array storing the input potential (at the current step)
//                                    --> _New: to be distinguishable from g_Pot_Array_USG[], which is defined at the previous step
//                g_Pot_Array_USG   : Array storing the input potential for UNSPLIT_GRAVITY (at the previous step)
//                g_Corner_Array    : Array storing the physical corner coordinates of each patch
//                dh                : Cell size
//                Undo              : Add (true) or subtract (false) potential correction to the input potential
//                USG               : Flag to indicate which potential is input
//
//-------------------------------------------------------------------------------------------------------
__global__
void CUPOT_CorrectEffPot(       real   g_Pot_Array_New[][ CUBE(GRA_NXT) ],
                                real   g_Pot_Array_USG[][ CUBE(USG_NXT_G) ],
                          const double g_Corner_Array [][3],
                          const real dh, const bool Undo, const bool USG, const int IDX, const int IDX_GZ )
{

   int IDX_sqr = SQR (IDX);

// load potential from global to shared memory to improve the GPU performance
   __shared__ real s_pot_new[ CUBE(GRA_NXT) ];
   __shared__ real s_pot_old[ CUBE(USG_NXT_G) ];

#  ifdef UNSPLIT_GRAVITY
   if ( USG )
      for (int t=threadIdx.x; t<CUBE(USG_NXT_G); t+=GRA_BLOCK_SIZE)
         s_pot_old[t] = g_Pot_Array_USG[blockIdx.x][t];
   else
      for (int t=threadIdx.x; t<CUBE(GRA_NXT); t+=GRA_BLOCK_SIZE)
         s_pot_new[t] = g_Pot_Array_New[blockIdx.x][t];
#  else
      for (int t=threadIdx.x; t<CUBE(GRA_NXT); t+=GRA_BLOCK_SIZE)
         s_pot_new[t] = g_Pot_Array_New[blockIdx.x][t];
#  endif

   __syncthreads();


// loop over all patches
   const int P = blockIdx.x;

   {
//    point to the potential array of the target patch
      real *pot;
//      const real *const pot = s_pot;
#     ifdef UNSPLIT_GRAVITY
         if ( USG )  pot = s_pot_old;
         else        pot = s_pot_new;
#     else
                     pot = s_pot_new;
#     endif


//    loop over all cells of the target patch
//    _g0: indices for the arrays without any ghost zone
      CGPU_LOOP( t, CUBE(IDX) )
      {

         const int i_g0 = t % IDX;
         const int j_g0 = t % IDX_sqr / IDX;
         const int k_g0 = t / IDX_sqr;

         const double dx = g_Corner_Array[P][0] + (double)((i_g0-IDX_GZ)*dh) - c_GREP_Center[0];
         const double dy = g_Corner_Array[P][1] + (double)((j_g0-IDX_GZ)*dh) - c_GREP_Center[1];
         const double dz = g_Corner_Array[P][2] + (double)((k_g0-IDX_GZ)*dh) - c_GREP_Center[2];

         const double r2 = SQR(dx) + SQR(dy) + SQR(dz);


         if ( r2 < c_r_max2 )
         {
            const double r = SQRT( r2 );

//          use binary search algorithm to find the index of bin
            int bin;
            for ( int i=0, j=c_GREP_NBin; j - i != 1; bin = (i + j) / 2 )
            {
               int mid = (i + j) / 2;
               if ( r > c_GREP_Edge[mid] )   i = mid;
               else                          j = mid;
            }

            double phi = ( bin == c_GREP_NBin-1 ) ? c_GREP_Data[bin]
                                                  : LinearInterp( r, c_GREP_Edge[bin], c_GREP_Edge[bin+1],
                                                                     c_GREP_Data[bin], c_GREP_Data[bin+1] );

//CHECK       some cells are outside the outermost EdgeR (Edge[bin + 1]), but why?
//            if ( (r < c_GREP_Edge[bin])  || ( r > c_GREP_Edge[bin + 1]) )
//            printf( "Incorrect index of bin %d for radius %.6e, EdgeL = %.6e and EdgeR = %.6e\n",
//                    bin, r, c_GREP_Edge[bin], c_GREP_Edge[bin+1]);

            if ( Undo )  pot[t] -= (real)phi;
            else         pot[t] += (real)phi;
         } // if ( r2 < r_max2 )

      } // CGPU_LOOP( idx_g0, CUBE(PS1) )
   }

} // FUNCTION : CPU_CorrectEffPot


#endif // #if ( defined GRAVITY  &&  defined GREP )
